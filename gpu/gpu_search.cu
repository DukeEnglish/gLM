#include "hip/hip_runtime.h"
#include "gpu_search.hh" 
//#include "entry_structs.hh"
#include <hip/hip_runtime.h>

#define MAX_NUM_CHILDREN 256
#define ENTRIES_PER_NODE (MAX_NUM_CHILDREN - 1)
#define ENTRY_SIZE (sizeof(unsigned int) + sizeof(unsigned int) + 2*sizeof(float)) //Same as the getEntrySize(true)
#define MAX_NGRAM 5
//Assume working with 256 thread DON'T RELY ENTIRERLY ON THOSE! Size may be smaller. need a parameter.
//Requires two more threads then num of entries per node


//We want to copy a whole BTree node to shared memory. We will know the size in advance, we need to distribute the copying between
//our threads. We might end up copying more than we need, but that is fine, as long as we avoid warp divergence.
__global__ void gpuSearchBtree(unsigned char * global_mem, unsigned int * keys, unsigned int * results){

    __shared__ unsigned int offsets[MAX_NUM_CHILDREN/2 +1]; //Reads in the first child offset + the shorts
    __shared__ unsigned int entries[ENTRIES_PER_NODE];
    __shared__ unsigned int prefix_sum; //Prefix sum gives us next node size
    __shared__ unsigned int found_idx;
    __shared__ unsigned int booleans[2]; //booleans[0] = is_last; booleans[1] = exact_match
    __shared__ unsigned int payload[3]; //After we find the correct entry, load the payload here
    __shared__ unsigned int keys_shared[MAX_NGRAM]; //Each block fetches from shared memory the max necessary number of keys

    //Maybe we need to issue shared memory here to optimize it
    int i = threadIdx.x;
    if (i < MAX_NGRAM) {
       keys_shared[i] = keys[(blockIdx.x*MAX_NGRAM) + i]; //Shared memory read here for up NUM_NGRAM keys 
    }
    __syncthreads();

    //Split some of the shared memory onto more comfrotable places
    unsigned short * offests_incremental = (unsigned short *)&offsets[1];
    unsigned int * first_child_offset = &offsets[0];

    unsigned int * is_last = &booleans[0];
    unsigned int * exact_match = &booleans[1];

    unsigned int * next_level = &payload[0];
    float * prob = (float *)&payload[1];
    float * backoff = (float *)&payload[2];

    int num_entries; //Set the number of entries per node depending on whether we are internal or leaf.

    //Set the start index
    unsigned int current_btree_start = 0;
    unsigned int current_ngram = 0;
    unsigned int key = keys_shared[current_ngram];
    while (key != 0 && current_ngram < MAX_NGRAM) {
        current_ngram++;
        unsigned int updated_idx = current_btree_start + 4; //Update the index for the while loop
        unsigned int size = *(unsigned int *)&global_mem[current_btree_start];; //The size of the current node to process. 
        //Move to register to avoid sychronizationIs it better to do this in shared memory

        //Initialize shared variable
        if (i < 2) {
            booleans[i] = false;
        }
        __syncthreads();

        while (!*exact_match) {
            //First warp divergence here. We are reading in from global memory
            if (i == 0) {
                //@TODO: Replace this with a mod check
                int cur_node_entries = (size - sizeof(unsigned int) - sizeof(unsigned short))/(ENTRY_SIZE + sizeof(unsigned short));
                *is_last = !(ENTRIES_PER_NODE == cur_node_entries);
                //@TODO. Fix this to be more efficient. Maybe move it with entries?
                //As per cuda memory model at least one write will succeed. We are clearing this value
                //So it doesn't interfere with the future values
                prefix_sum = 0;
            }
            __syncthreads();


            if (*is_last) {
                //The number of entries in the bottom most nodes may be smaller than the size
                num_entries = size/ENTRY_SIZE;
                if (i < num_entries) {
                    entries[i] = *(unsigned int *)(&global_mem[updated_idx + i*sizeof(unsigned int)]);
                    //printf("Entries i: %d, value %d\n", i, entries[i]);
                }
                //printf("Num entries: %d size: %d\n", num_entries, size);
            } else {
                num_entries = ENTRIES_PER_NODE;
                //Load the unsigned int start offset together with the accumulated offsets to avoid warp divergence
                if (i < (MAX_NUM_CHILDREN/2) + 1) {
                    offsets[i] = *(unsigned int *)(&global_mem[updated_idx + i*sizeof(unsigned int)]);
                }
                //Now load the entries
                if (i < num_entries) {
                    entries[i] = *(unsigned int *)(&global_mem[updated_idx + sizeof(unsigned int) + MAX_NUM_CHILDREN*sizeof(unsigned short) + i*sizeof(unsigned int)]);
                }
            }
            __syncthreads();

            //NOW search
            if (i == 0) {
                if (key <= entries[i]) {
                    found_idx = i;
                    if (key == entries[i]) {
                        *exact_match = true;
                    }
                }
            } else if (i < num_entries) {
                if (key > entries[i-1] && key <= entries[i]){
                    found_idx = i;
                    if (key == entries[i]) {
                        *exact_match = true;
                    }
                }
            } else if (i == num_entries) {
                //Case where our key is greater than the last available entry. We need to do a prefix sum of i+1 elements.
                if (key > entries[i-1]) {
                    found_idx = i;
                }
            }
            __syncthreads();

            //We found either an exact match (so we can access next level) or at least an address to next btree level
            if (!*exact_match && !*is_last) {
                //Do a prefix sum on the offsets here
                //@TODO optimize later. Do a proper prefix sum rather than atomic add
                if (i < found_idx) {
                   atomicAdd(&prefix_sum, (int)offests_incremental[i]); 
                }
                __syncthreads(); //This is not necessary? It is necssary because the threads that don't take the if
                //path may write to the updated idx
                //As per the cuda memory model at least one write will succeed. since they all write the same we don't care
                size = (int)offests_incremental[found_idx];
                updated_idx = *first_child_offset + prefix_sum;
                __syncthreads(); //Data hazard fix on size
                
            } else if (*is_last && !*exact_match) {
                //In this case we didn't find the key that we were looking for
                //@TODO return a invalid offset when we didn't find anything (mb 0)?
                if (i == 0) {
                    //printf("Key not found! Key was %d\n", key);
                    results[blockIdx.x*3] = 0; //Indicate that we didn't find the key that we were looking for
                }

                break;

            } else {
                //Locate the rest of the data for the entry (i.e. the payload - backoff, prob, next offset)
                if (i < 3) {
                    //What we are doing here is reading the correct memory location for our payload. The payload is found
                    //After the offsets and the keys, so we skip them and then we skip to the correct payload using found_idx
                    if (*is_last) {
                        payload[i] = *(unsigned int *)(&global_mem[updated_idx + num_entries*sizeof(unsigned int) //Skip the keys
                            + found_idx*(sizeof(unsigned int) + sizeof(float) + sizeof(float)) //Skip the previous keys' payload
                                + i*sizeof(unsigned int)]); //Get next_level/prob/backoff
                    } else {
                        payload[i] = *(unsigned int *)(&global_mem[updated_idx + sizeof(unsigned int) + MAX_NUM_CHILDREN*sizeof(unsigned short) //Skip the offsets and first offset
                            + num_entries*sizeof(unsigned int) //Skip the keys
                                + found_idx*(sizeof(unsigned int) + sizeof(float) + sizeof(float)) //Skip the previous keys' payload
                                    + i*sizeof(unsigned int)]);  //Get next_level/prob/backoff
                    }
                    results[blockIdx.x*3 + i] = payload[i]; //Copy the results of the search to an array which will later be checked
                }

                key = keys_shared[current_ngram];
                if (current_ngram < MAX_NGRAM && key != 0) {
                    __syncthreads();
                    current_btree_start = *next_level;
                    //if (i == 0) {
                    //    printf("Current_btree_start: %d current_ngram: %d\n", *next_level, current_ngram);
                    //}
                }
                
                //if (i == 1) {
                //    printf("Exact match! Found_idx: %d, key: %d found: %d\nNext level: %d, prob %f, backoff %f\n", found_idx, key, entries[found_idx], *next_level, *prob, *backoff);
                //}
                break;
            }
        }
    }
}

void searchWrapper(unsigned char * global_mem, unsigned int * keys, unsigned int num_keys, unsigned int * results) {
    //Block size should always be MAX_NUM_CHILDREN for best efficiency when searching the btree
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    gpuSearchBtree<<<num_keys, MAX_NUM_CHILDREN>>>(global_mem, keys, results);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Searched %d keys in: %f milliseconds.\n", num_keys, milliseconds);
}

/* Can't compile easily with cmake. Maybe there's a better way
__global__ void searchInBulk(unsigned int * keys_array, unsigned char * btree_trie) {
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    gpuSearchBtree<<<1, MAX_NUM_CHILDREN>>>(btree_trie, 0, keys_array[i]);
}
*/

void cudaDevSync() {
    hipDeviceSynchronize();
}
